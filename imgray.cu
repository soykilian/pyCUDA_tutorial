
#include <hip/hip_runtime.h>
__global__ void applyFilter(unsigned char *redChannel,
                            unsigned char *greenChannel,
                            unsigned char *blueChannel,
                            const unsigned int width, const unsigned int height) {
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < height && col < width) {
        const unsigned int index = col + row * width;
        const unsigned char intensity = static_cast<unsigned char>(
            redChannel[index] * 0.2126 + greenChannel[index] * 0.7152 + blueChannel[index] * 0.0722
        );
        redChannel[index] = intensity;
        greenChannel[index] = intensity;
        blueChannel[index] = intensity;
    }
}
